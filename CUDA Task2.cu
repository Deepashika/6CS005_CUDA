#include "hip/hip_runtime.h"
// Name: Deepashika Maduwanthi
// Student Id: 1432291
//--------------------------------------------------

/*include <stdlib.h> header provides variable types,several macros, 
and functions to performe general functions.

 include <stdio.h> 
-the compiler to include this header file for compilation
*/


#include<stdio.h>
#include <stdlib.h>


#define N 4

/*Global function is also called "kernels".
 It's the functions that you may call from the host side.
 Matri_Add()will execute on the device
Matri_Add()will be called from the host
*/

__global__ void Matri_Add(int A[][N], int B[][N], int C[][N]){
	 // Thread row and column 
        int i = threadIdx.x;
		int j = threadIdx.y;

		C[i][j] = A [i][j] + B[i][j];
}


int main()
{

int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };
   

//calling the poniters
int (*d_A)[N], (*d_B)[N], (*d_C)[N];

// allocate device copies of A,B, C	
	hipMalloc((void**)&d_A, (N*N)*sizeof(int));
	hipMalloc((void**)&d_B, (N*N)*sizeof(int));
	hipMalloc((void**)&d_C, (N*N)*sizeof(int));
	
// CUDA memory copy types(copy input to device from host)
  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

   int numBlocks = 1;
  
  // N threads (kernel invoke N threads)
  dim3 threadsPerBlock(N,N);
  Matri_Add<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

  // copy result of device back to host 
  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

	int i, j; printf("C = \n"); 
	for(i=0;i<N;i++){
	for(j=0;j<N;j++){ 
	printf("%d ", C[i][j]);
	}
	printf("\n");
}
  