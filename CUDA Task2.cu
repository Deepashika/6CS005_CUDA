// Name: Deepashika Maduwanthi
// Student Id: 1432291
//--------------------------------------------------

/*include <stdlib.h> header provides variable types,several macros, 
and functions to performe general functions.

 include <stdio.h> 
-the compiler to include this header file for compilation
*/



#include <hip/hip_runtime.h>
#include<stdio.h>
#include <stdlib.h>


#define N 4

/*Global function is also called "kernels".
 It's the functions that you may call from the host side.
 Matri_Add()will execute on the device
Matri_Add()will be called from the host
*/

__global__ void Matri_Add(int A[][N], int B[][N], int C[][N]){
	 // Thread row and column 
        int i = threadIdx.x;
		int j = threadIdx.y;

		C[i][j] = A [i][j] + B[i][j];
}


int main()
{
int i, j =0;

int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };


for(i=0;i<N;i++){
for(j=0;j<N;j++){
C[i][j] = A[i][j] + B[i][j];
        }
    }

printf("Sum of entered matrices: \n");

for(i=0;i<N;i++){
for(j=0;j<N;j++){
printf("%d ", C[i][j]);
        }
printf("\n");
    }

return 0;
}
