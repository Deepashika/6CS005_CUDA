#include "hip/hip_runtime.h"
// Name: Deepashika Maduwanthi
// Student Id: 1432291
//--------------------------------------------------

/*include <stdlib.h> header provides variable types,several macros, 
and functions to performe general functions.

 include <stdio.h> 
-the compiler to include this header file for compilation
*/


#include<stdio.h>
#include <stdlib.h>


#define N 4

/*Global function is also called "kernels".
 It's the functions that you may call from the host side.
 Matri_Add()will execute on the device
Matri_Add()will be called from the host
*/

__global__ void Matri_Add(int A[][N], int B[][N], int C[][N]){
	 // Thread row and column 
        int i = threadIdx.x;
		int j = threadIdx.y;

		C[i][j] = A [i][j] + B[i][j];
}


int main()
{

int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };
   

//calling the poniters
int (*d_A)[N], (*d_B)[N], (*d_C)[N];

// allocate device copies of A,B, C	
	hipMalloc((void**)&d_A, (N*N)*sizeof(int));
	hipMalloc((void**)&d_B, (N*N)*sizeof(int));
	hipMalloc((void**)&d_C, (N*N)*sizeof(int));