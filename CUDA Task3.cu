#include "hip/hip_runtime.h"
// Name: Deepashika Maduwanthi
// Student Id: 1432291
//--------------------------------------------------

/modify the CUDA_task2 program to generate A and B matrix automatically/

/*These are header files, include <stdio.h> 
-the compiler to include this header file for compilation
stdlib.h header provides variable types,several macros, 
and functions to performe general functions.*/

#include <stdio.h>
#include <stdlib.h>

#define N 4


/*Global function is also called "kernels".
 It's the functions that you may call from the host side.
 Matri_Add()will execute on the device
Matri_Add()will be called from the host
*/
  
/*Global function is also called "kernels".
 It's the functions that you may call from the host side.
 
*/

__global__ void Matri_Add(int A[][N], int B[][N], int C[][N]){
	
	
	 // Thread row and column 
        int i = threadIdx.x;
		int j = threadIdx.y;

		C[i][j] = A [i][j] + B[i][j];

}

int randmatfunc(){
  int i, j, k; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          k = rand() % 100 + 1;;
            printf("%d ", k);
            newmat[i][j] =k;
        }
        printf("\n");
       
    } 
  printf("\n-----------------------------------\n"); 
}

// main function of the program

int main(){

int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };

//calling the poniters
  int (*d_A)[N], (*d_B)[N], (*d_C)[N];

// allocate device copies of A,B, C	
  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

 // CUDA memory copy types(copy input to device from host)
  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = 1;
  
  // N threads (kernel invoke N threads)
  dim3 threadsPerBlock(N,N);
  Matri_Add<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

// copy result of device back to host 
  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

	int i, j; printf("C = \n"); 
	for(i=0;i<N;i++){
	for(j=0;j<N;j++){ 
	printf("%d ", C[i][j]);
	}
	printf("\n");
	}

//  cleanup 
  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}